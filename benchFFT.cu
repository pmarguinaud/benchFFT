
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <stdio.h>
#include <stdlib.h>

#define cufftSafeCall(err) __cufftSafeCall (err, __FILE__, __LINE__)

static const char *_cudaGetErrorEnum (hipfftResult error)
{
  switch (error)
    {
#define cr(x) case CUFFT_##x: return #x
      cr (SUCCESS);
      cr (INVALID_PLAN);
      cr (ALLOC_FAILED);
      cr (INVALID_TYPE);
      cr (INVALID_VALUE);
      cr (INTERNAL_ERROR);
      cr (EXEC_FAILED);
      cr (SETUP_FAILED);
      cr (INVALID_SIZE);
      cr (UNALIGNED_DATA);
#undef cr
    }
  return "UNKNOWN";
}

inline void __cufftSafeCall (hipfftResult err, const char * file, const int line)
{
  if (HIPFFT_SUCCESS != err) 
    {
      fprintf (stderr, "CUFFT error in file '%s'\n",__FILE__);
      fprintf (stderr, "CUFFT error %d: %s\nterminating!\n", err, _cudaGetErrorEnum (err)); 
      hipDeviceReset (); 
    }
}


int main (int argc, char * argv[])
{

  if (argc < 8)
    {
      fprintf (stderr, "Usage: %s N LOT istride ostride idist odist\n", argv[0]);
      return 1;
    }

  int N       = atoi (argv[1]);
  int LOT     = atoi (argv[2]);
  int istride = atoi (argv[3]);
  int ostride = atoi (argv[4]);
  int idist   = atoi (argv[5]);
  int odist   = atoi (argv[6]);
  int llprint = atoi (argv[7]);

  hipfftHandle plan;

  if (hipDeviceSynchronize() != hipSuccess)
    {
      fprintf(stderr, "Cuda error: Failed to synchronize\n");
      return 1;	
    }

  int embed[1] = {1};

  cufftSafeCall (hipfftCreate (&plan));

  cufftSafeCall (hipfftPlanMany (&plan, 1, &N, embed, istride, idist, embed, ostride, odist, HIPFFT_D2Z, LOT));

  if (llprint)
  printf (" N = %d, LOT = %d, istride = %d, ostride = %d, idist = %d, odist = %d\n", N, LOT, istride, ostride, idist, odist);

  if (hipDeviceSynchronize () != hipSuccess)
    {
      fprintf(stderr, "Cuda error: Failed to synchronize\n");
      return 1;	
    }

  double * z = (double *)malloc (sizeof (double) * LOT * idist);

  for (int j = 0; j < LOT; j++)
  for (int i = 0; i < idist; i++)
    z[j*idist+i] = (i >= N) ? 9999. : (i %2) ? +1. : -1.;

  if (llprint)
  for (int j = 0; j < LOT; j++)
    {
      for (int i = 0; i < idist; i++)
        printf (" %8.1f", z[j*idist+i]);
      printf ("\n");
    }

  hipfftDoubleComplex * data = NULL;

  size_t sz = sizeof (double) * LOT * idist;

  hipMalloc ((void**)&data, sz);

  hipMemcpy (data, z, sz, hipMemcpyHostToDevice);

  cufftSafeCall (hipfftExecD2Z (plan, (hipfftDoubleReal*)data, data));

  hipMemcpy (z, data, sz, hipMemcpyDeviceToHost);

  if (llprint)
  for (int j = 0; j < LOT; j++)
    {
      for (int i = 0; i < idist; i++)
        printf (" %8.1f", z[j*idist+i]);
      printf ("\n");
    }


  return 0;
}
