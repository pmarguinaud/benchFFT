
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <time.h>

#define cufftSafeCall(err) __cufftSafeCall (err, __FILE__, __LINE__)

static const char *_cudaGetErrorEnum (hipfftResult error)
{
  switch (error)
    {
#define cr(x) case CUFFT_##x: return #x
      cr (SUCCESS);
      cr (INVALID_PLAN);
      cr (ALLOC_FAILED);
      cr (INVALID_TYPE);
      cr (INVALID_VALUE);
      cr (INTERNAL_ERROR);
      cr (EXEC_FAILED);
      cr (SETUP_FAILED);
      cr (INVALID_SIZE);
      cr (UNALIGNED_DATA);
#undef cr
    }
  return "UNKNOWN";
}

inline void __cufftSafeCall (hipfftResult err, const char * file, const int line)
{
  if (HIPFFT_SUCCESS != err) 
    {
      fprintf (stderr, "CUFFT error in file '%s'\n",__FILE__);
      fprintf (stderr, "CUFFT error %d: %s\nterminating!\n", err, _cudaGetErrorEnum (err)); 
      hipDeviceReset (); 
    }
}


int main (int argc, char * argv[])
{

  if (argc < 10)
    {
      fprintf (stderr, "Usage: %s N LOT istride ostride idist odist llprint kfunc ntime\n", argv[0]);
      return 1;
    }

  int N       = atoi (argv[1]);
  int LOT     = atoi (argv[2]);
  int istride = atoi (argv[3]);
  int ostride = atoi (argv[4]);
  int idist   = atoi (argv[5]);
  int odist   = atoi (argv[6]);
  int llprint = atoi (argv[7]);
  int kfunc   = atoi (argv[8]);
  int ntime   = atoi (argv[9]);

  assert ((istride == 1) || (idist == 1));
  assert ((ostride == 1) || (odist == 1));

  hipfftHandle plan;

  if (hipDeviceSynchronize() != hipSuccess)
    {
      fprintf(stderr, "Cuda error: Failed to synchronize\n");
      return 1;	
    }

  int embed[1] = {1};

  cufftSafeCall (hipfftCreate (&plan));

  cufftSafeCall (hipfftPlanMany (&plan, 1, &N, embed, istride, idist, embed, ostride, odist, HIPFFT_D2Z, LOT));

  if (llprint)
  printf (" N = %d, LOT = %d, istride = %d, ostride = %d, idist = %d, odist = %d\n", N, LOT, istride, ostride, idist, odist);

  if (hipDeviceSynchronize () != hipSuccess)
    {
      fprintf(stderr, "Cuda error: Failed to synchronize\n");
      return 1;	
    }

  size_t sz = LOT * idist + N * istride + 2 * LOT;

  if (llprint)
    printf (" sz = %ld\n", sz);

  double * z = (double *)malloc (sz * sizeof (double));

  for (int i = 0; i < sz; i++)
    z[i] = 9999.;


  for (int j = 0; j < LOT; j++)
  for (int i = 0; i < N; i++)
    {
      double zval = 0.;
      switch (kfunc)
        {
          case 1: zval = (i % 4) ? +1. : -1.; break;
          case 2: zval = (i % 2) ? +1. : -1.; break;
          default: zval = 1.;
        }
      z[j*idist+i*istride] = zval;
    }


  if (llprint == 1)
  for (int j = 0; j < LOT; j++)
    {
      for (int i = 0; i < N+2; i++)
        printf (" %8.1f", z[j*idist+i*istride]);
      printf ("\n");
    }

  if (llprint == 2)
  for (int i = 0; i < sz; i++)
    {
      printf (" %8.1f", z[i]);
      if ((((i + 1) % 20) == 0) || (i == sz - 1)) printf ("\n");
    }

  hipfftDoubleComplex * data = NULL;

  hipMalloc ((void**)&data, sz * sizeof (double));

  hipMemcpy (data, z, sz * sizeof (double), hipMemcpyHostToDevice);


  clock_t t0 = clock ();
  for (int itime = 0; itime < ntime; itime++)
    cufftSafeCall (hipfftExecD2Z (plan, (hipfftDoubleReal*)data, data));
  clock_t t1 = clock ();

  printf (" sz = %ld, dt = %f\n", sz, (double)(t1-t0)/1e+6);

  hipMemcpy (z, data, sz * sizeof (double), hipMemcpyDeviceToHost);

  if (llprint == 1)
  for (int j = 0; j < LOT; j++)
    {
      for (int i = 0; i < N+2; i++)
        printf (" %8.1f", z[j*idist+i*istride]);
      printf ("\n");
    }

  if (llprint == 2)
  for (int i = 0; i < sz; i++)
    {
      printf (" %8.1f", z[i]);
      if ((((i + 1) % 20) == 0) || (i == sz - 1)) printf ("\n");
    }


  return 0;
}
